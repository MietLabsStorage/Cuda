#include "hip/hip_runtime.h"
﻿#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include <cstdio>

#define N (1024*1024)

__global__ void kernel(float* data)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const float x = 2 * static_cast<float>(3.1415926) * static_cast<float>(idx) / static_cast<float>(N);
	data[idx] = sinf(sqrtf(x));
}

void hello_world()
{
	const auto a = static_cast<float*>(malloc(N * sizeof(float)));
	float* dev = nullptr;
	// выделить память на GPU
	hipMalloc(reinterpret_cast<void**>(&dev), N * sizeof(float));
	// конфигурация запуска N нитей
	kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);
	// скопировать результаты в память CPU
	hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);
	// освободить выделенную память
	hipFree(dev);

	for (int idx = 0; idx < N; idx++)
		printf("a[%d] = %.5f\n", idx, a[idx]);

	free(a);
}

void about_devices()
{
	int device_count;
	hipDeviceProp_t dev_prop{};
	hipGetDeviceCount(&device_count);
	printf("Found %d devices\n", device_count);
	for (int device = 0; device < device_count; device++)
	{
		hipGetDeviceProperties(&dev_prop, device);
		printf("Device %d\n", device);
		printf("Compute capability : %d.%d\n", dev_prop.major, dev_prop.minor);
		printf("Name : %s\n", dev_prop.name);
		printf("Total Global Memory : %llu\n", dev_prop.totalGlobalMem);
		printf("Shared memory per block: %llu\n", dev_prop.sharedMemPerBlock);
		printf("Registers per block : %d\n", dev_prop.regsPerBlock);
		printf("Warp size : %d\n", dev_prop.warpSize);
		printf("Max threads per block : %d\n", dev_prop.maxThreadsPerBlock);
		printf("Max threads dim : %d*%d*%d\n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
		printf("Total constant memory : %llu\n", dev_prop.totalConstMem);
		printf("SM count: %d\n", dev_prop.multiProcessorCount);
	}
}

void time_tracker(bool default_blocks_count)
{
	//описываем переменные типа hipEvent_t
	hipEvent_t start, stop;

	float gpu_time = 0.0f;

	// создаем события начала и окончания выполнения ядра
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* dev = nullptr;
	hipMalloc(reinterpret_cast<void**>(&dev), N * sizeof(float));

	//привязываем событие start к данному месту
	hipEventRecord(start, nullptr);
	// вызвать ядро
	if (default_blocks_count)
	{
		kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);
	}
	else
	{
		kernel << <dim3(16, 1), dim3(512, 1) >> > (dev);
	}
	//привязываем событие stop к данному месту
	hipEventRecord(stop, nullptr);
	hipEventSynchronize(stop);

	hipFree(dev);

	// запрашиваем время между событиями
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("time spent executing by the GPU (%d blocks): %.5f ms\n", default_blocks_count ? (N / 512) : 16, static_cast<double>(gpu_time));
	// уничтожаем созданные события
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

int main(int argc, char* argv[])
{
	about_devices();

	system("pause");

	time_tracker(true);
	time_tracker(false);

	system("pause");

	hello_world();

	return 0;
}