#include "hip/hip_runtime.h"
﻿#include "helper_image.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>

using namespace std;

texture<unsigned char, 2, hipReadModeElementType> g_Bilinear;
__global__ void Bilinear_kernel(unsigned char* pDst,
    float factor,
    int w, int h)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidx < w && tidy < h)
    {
        float center = tidx / factor;
        int start = (int)center;
        int stop = start + 1.0f;
        float t = center - start;
        float a = tex2D(g_Bilinear, tidy + 0.5f, start + 0.5f);
        float b = tex2D(g_Bilinear, tidy + 0.5f, stop + 0.5f);
        float linear = a + t*(b-a);
        pDst[tidx + tidy * w] = (int)linear;
    }
}

void loadImage(char* file, unsigned char** pixels, unsigned int* width, unsigned int* height)
{
    size_t file_length = strlen(file);
    if (!strcmp(&file[file_length - 3], "pgm"))
    {
        if (sdkLoadPGM<unsigned char>(file, pixels, width, height) != true)
        {
            printf("Failed to load PGM image file: %s\n", file);
            exit(EXIT_FAILURE);
        }
    }
}
void saveImage(char* file, unsigned char* pixels, unsigned int width, unsigned int height)
{
    size_t file_length = strlen(file);
    if (!strcmp(&file[file_length - 3], "pgm"))
    {
        sdkSavePGM(file, pixels, width, height);
    }
}

unsigned int width = 512, height = 512;

void rotate(char* name_in, char* name_out)
{
    unsigned char* d_result_pixels;
    unsigned char* h_result_pixels;
    unsigned char* h_pixels = NULL;
    unsigned char* d_pixels = NULL;

    char* src_path = name_in;//"mj.pgm";
    char* d_result_path = name_out;// "mj_d.pgm";
    loadImage(src_path, &h_pixels, &width, &height);
    int image_size = sizeof(unsigned char) * width * height;
    h_result_pixels = (unsigned char*)malloc(image_size);
    hipMalloc((void**)&d_pixels, image_size);
    hipMalloc((void**)&d_result_pixels, image_size);
    hipMemcpy(d_pixels, h_pixels, image_size, hipMemcpyHostToDevice);
    int n = 16;
    dim3 block(n, n);
    dim3 grid(width / n, height / n);
    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar1>();
    size_t offset = 0;
    hipError_t error = hipBindTexture2D(0, &g_Bilinear, d_pixels, &desc, width, height, width
        * sizeof(unsigned char));
    if (hipSuccess != error) {
        printf("ERROR: Failed to bind texture.\n");
        exit(-1);
    }
    else {
        printf("Texture was successfully binded\n");
    }
    Bilinear_kernel << < grid, block >> > (d_result_pixels, 2, width, height );
    hipMemcpy(h_result_pixels, d_result_pixels, image_size, hipMemcpyDeviceToHost);

    saveImage(d_result_path, h_result_pixels, width, height);
    hipUnbindTexture(&g_Bilinear);
}

int main()
{
    rotate("mj.pgm", "mj1.pgm");
    rotate("mj1.pgm", "mj2.pgm");
    printf("DONE\n");

    return 0;
}
